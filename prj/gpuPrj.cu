#include "hip/hip_runtime.h"
/*
 *   Description: use GPU to calculate parallel beam, fan beam projection 
 *   and their corresponding back projection.
 *
 *   Reference:
 *   Author: Renliang Gu (renliang@iastate.edu)
 *   $Revision: 0.1 $ $Date: Sun 10 Nov 2013 01:23:32 AM CST
 *
 *   v_0.1:     first draft
 */
#define GPU 1

/*#include <unistd.h>*/
#include <stdlib.h>
#include <math.h>
#include <stdio.h>
#include <time.h>
#include <pthread.h>
#include <limits.h>
#include <stdio.h>
#include <stddef.h>
#include <string.h>
#include "./common/kiss_fft.h"

#ifdef __cplusplus
extern "C"{
#endif
#include "prj.h"
#ifdef __cplusplus
}
#endif
#include "utils.h"

#if EXE_PROF
#if GPU
#include <hip/hip_runtime_api.h>
#endif
#endif

#if SHOWIMG
#include "./common/cpu_bitmap.h"
#endif

struct prjConf config;
struct prjConf* pConf = &config;

#if GPU
hipEvent_t     start, stop;

ft *dev_img;
ft *dev_sino;

static dim3 fGrid, fThread, bGrid, bThread;

__constant__ prjConf dConf;
#endif

#if GPU
/*
   */
static void HandleError( hipError_t err,
                         const char *file,
                         int line ) {
    if (err != hipSuccess) {
        printf( "%s in %s at line %d\n", hipGetErrorString( err ),
                file, line );
        exit( EXIT_FAILURE );
    }
}
#endif

__global__ void pixelDrivePar(ft* img, ft* sino,int FBP){
    // detector array is of odd size with the center at the middle
    ft theta;       // the current angle, range in [0 45]
    int thetaIdx; // index counts from 0
    prjConf* conf = &dConf;

    int pC = conf->prjWidth/2;
    int N=conf->n;

    int tileSz = sqrtf(blockDim.x);
    int tileX=0, tileY=blockIdx.x;
    while(tileY>tileX){ tileX++; tileY-=tileX; }

    int xl=tileX*tileSz, yt=tileY*tileSz;

    int x=xl+threadIdx.x/tileSz,
        y=yt+threadIdx.x%tileSz;

    ft cosT, sinT;  // cosine and sine of theta
    ft tileSzSinT, tileSzCosT;
#if DEBUG
    if(blockIdx.x==0 && threadIdx.x==0)
        printf("entering pixelDrive grid=(%d,%d), block(%d,%d), blockIdx=(%d,%d), threadIdx=(%d,%d), (%d,%d)\n",
                gridDim.x,gridDim.y,blockDim.x,blockDim.y,blockIdx.x,blockIdx.y,threadIdx.x,threadIdx.y,
                x,y);
#endif


    // for each point (x,y) on the ray is
    // x= t*cosT + (t*cosT+d*sinT)/(t*sinT-d*cosT)*(y-t*sinT);
    // or x = -t*d/(t*sinT-d*cosT) + y*(t*cosT+d*sinT)/(t*sinT-d*cosT);
    ft oc;
    ft beamWidth = conf->dSize*conf->effectiveRate;

    ft dist, weight;
    int temp, imgIdx;
    ft imgt[8];
    ft t, tl, tr, tll, trr;
    int dt, dtl, dtr, dtll, dtrr;
    for(int i=0; i<8; i++) imgt[i]=0;
    __shared__ volatile ft shared[8][2][4*TILE_SZ];
    for(thetaIdx=0; thetaIdx<conf->prjFull/2;thetaIdx++){

        theta  = thetaIdx *2*PI/conf->prjFull;
        cosT = cos(theta ); sinT = sin(theta );
        tileSzCosT=tileSz*cosT; tileSzSinT=tileSz*sinT;

        // up letf
        oc = (xl-0.5)*cosT + (yt-0.5)*sinT; tll = oc; trr = tll;

        // up right
        //qe = (xr+0.5)*sinTl - (yt-0.5)*cosTl + d;
        //oc = (xr+0.5)*cosTl + (yt-0.5)*sinTl;
        oc = oc+tileSzCosT; t=oc; tll = min(tll, t); trr=max(trr,t);

        // bottom right
        //qe = (xr+0.5)*sinTl - (yb+0.5)*cosTl + d;
        //oc = (xr+0.5)*cosTl + (yb+0.5)*sinTl;
        oc = oc+tileSzSinT; t=oc; tll = min(tll, t); trr=max(trr,t);

        // bottom left
        //qe = (xl-0.5)*sinTl - (yb+0.5)*cosTl + d;
        //oc = (xl-0.5)*cosTl + (yb+0.5)*sinTl;
        oc = oc-tileSzCosT; t=oc; tll = min(tll, t); trr=max(trr,t);

        // up letf
        oc = (x-0.5)*cosT + (y-0.5)*sinT;
        tl = oc; tr = tl;

        // up right
        //qe = (x+0.5)*sinT - (y-0.5)*cosT + d;
        //oc = (x+0.5)*cosT + (y-0.5)*sinT;
        oc = oc + cosT; t = oc;
        tl = min(tl, t); tr=max(tr,t);

        // bottom right
        //qe = (x+0.5)*sinT - (y+0.5)*cosT + d;
        //oc = (x+0.5)*cosT + (y+0.5)*sinT;
        oc=oc+sinT; t = oc;
        tl = min(tl, t); tr=max(tr,t);

        // bottom left
        //qe = (x-0.5)*sinT - (y+0.5)*cosT + d;
        //oc = (x-0.5)*cosT + (y+0.5)*sinT;
        oc = oc-cosT; t = oc;
        tl = min(tl, t); tr=max(tr,t);

        //qe = d+x*sinT-y*cosT; // positive direction is qo
        dtl = max((int)round((tl)/conf->dSize),-(conf->prjWidth-1)/2);
        dtr = min((int)round((tr)/conf->dSize), (conf->prjWidth-1)/2);

        dtll=max((int)round((tll)/conf->dSize),-(conf->prjWidth-1)/2);
        dtrr=min((int)round((trr)/conf->dSize), (conf->prjWidth-1)/2);

        dtl = max(dtl,dtll); dtr=min(dtr,dtrr);

        __syncthreads();
        for(dt=dtll+threadIdx.x; dt<=dtrr; dt+=blockDim.x){
            temp=thetaIdx;
            shared[0][0][dt-dtll]=temp<conf->np? sino[temp*conf->prjWidth+dt+pC] : 0;
            shared[0][1][dt-dtll]=temp<conf->np? sino[temp*conf->prjWidth-dt+pC] : 0;

            temp=(thetaIdx+conf->prjFull/4)%conf->prjFull;
            shared[1][0][dt-dtll]=temp<conf->np? sino[temp*conf->prjWidth+dt+pC] : 0;
            shared[1][1][dt-dtll]=temp<conf->np? sino[temp*conf->prjWidth-dt+pC] : 0;

            temp=(thetaIdx+conf->prjFull/2)%conf->prjFull;
            shared[2][0][dt-dtll]=temp<conf->np? sino[temp*conf->prjWidth+dt+pC] : 0;
            shared[2][1][dt-dtll]=temp<conf->np? sino[temp*conf->prjWidth-dt+pC] : 0;

            temp=(thetaIdx+3*conf->prjFull/4)%conf->prjFull;
            shared[3][0][dt-dtll]=temp<conf->np? sino[temp*conf->prjWidth+dt+pC] : 0;
            shared[3][1][dt-dtll]=temp<conf->np? sino[temp*conf->prjWidth-dt+pC] : 0;

            temp=(3*conf->prjFull/2-thetaIdx)%conf->prjFull;
            shared[4][0][dt-dtll]=temp<conf->np? sino[temp*conf->prjWidth-dt+pC] : 0;
            shared[4][1][dt-dtll]=temp<conf->np? sino[temp*conf->prjWidth+dt+pC] : 0;

            temp=(7*conf->prjFull/4-thetaIdx)%conf->prjFull;
            shared[5][0][dt-dtll]=temp<conf->np? sino[temp*conf->prjWidth-dt+pC] : 0;
            shared[5][1][dt-dtll]=temp<conf->np? sino[temp*conf->prjWidth+dt+pC] : 0;

            temp=(conf->prjFull-thetaIdx)%conf->prjFull;
            shared[6][0][dt-dtll]=temp<conf->np? sino[temp*conf->prjWidth-dt+pC] : 0;
            shared[6][1][dt-dtll]=temp<conf->np? sino[temp*conf->prjWidth+dt+pC] : 0;

            temp=(5*conf->prjFull/4-thetaIdx)%conf->prjFull;
            shared[7][0][dt-dtll]=temp<conf->np? sino[temp*conf->prjWidth-dt+pC] : 0;
            shared[7][1][dt-dtll]=temp<conf->np? sino[temp*conf->prjWidth+dt+pC] : 0;
        }
        __syncthreads();

        for(dt=dtl; dt<=dtr; dt++){
            t = dt*conf->dSize;
            dist=x*cosT+y*sinT-t;
            weight=getWeight(dist,beamWidth,cosT,sinT);

            imgt[0] += weight*shared[0][0][dt-dtll];
            imgt[2] += weight*shared[0][1][dt-dtll];
            imgt[1] += weight*shared[1][0][dt-dtll];
            imgt[3] += weight*shared[1][1][dt-dtll];
            imgt[2] += weight*shared[2][0][dt-dtll];
            imgt[0] += weight*shared[2][1][dt-dtll];
            imgt[3] += weight*shared[3][0][dt-dtll];
            imgt[1] += weight*shared[3][1][dt-dtll];
            imgt[4] += weight*shared[4][0][dt-dtll];
            imgt[6] += weight*shared[4][1][dt-dtll];
            imgt[5] += weight*shared[5][0][dt-dtll];
            imgt[7] += weight*shared[5][1][dt-dtll];
            imgt[6] += weight*shared[6][0][dt-dtll];
            imgt[4] += weight*shared[6][1][dt-dtll];
            imgt[7] += weight*shared[7][0][dt-dtll];
            imgt[5] += weight*shared[7][1][dt-dtll];
        }
    }
    if(FBP) for(int i=0; i<8; i++) imgt[i]=imgt[i]*PI/conf->np;
    if(x>(N-1)/2 || y>(N-1)/2) return;

    imgIdx = ( y+N/2)*N+x+N/2; img[imgIdx] = imgt[0]/conf->effectiveRate;
    imgIdx = ( x+N/2)*N-y+N/2; img[imgIdx] = imgt[1]/conf->effectiveRate;
    imgIdx = (-y+N/2)*N-x+N/2; img[imgIdx] = imgt[2]/conf->effectiveRate;
    imgIdx = (-x+N/2)*N+y+N/2; img[imgIdx] = imgt[3]/conf->effectiveRate;
    if(y==0 || y>=x) return;
    imgIdx = (-y+N/2)*N+x+N/2; img[imgIdx] = imgt[4]/conf->effectiveRate;
    imgIdx = ( x+N/2)*N+y+N/2; img[imgIdx] = imgt[5]/conf->effectiveRate;
    imgIdx = ( y+N/2)*N-x+N/2; img[imgIdx] = imgt[6]/conf->effectiveRate;
    imgIdx = (-x+N/2)*N-y+N/2; img[imgIdx] = imgt[7]/conf->effectiveRate;
}

__global__ void pixelDriveFan(ft* img, ft* sino, int FBP){
    // printf("entering pixelDrive\n");
    // detector array is of odd size with the center at the middle
    ft theta;       // the current angle, range in [0 45]
    int thetaIdx; // index counts from 0
    prjConf* conf = &dConf;

    int pC = conf->prjWidth/2;
    ft d=conf->d; // the distance from rotating center to source in pixel
    int N=conf->n;

    int tileSz = sqrtf(blockDim.x);
    int tileX=0, tileY=blockIdx.x;
    while(tileY>tileX){ tileX++; tileY-=tileX; }

    int xl=tileX*tileSz, yt=tileY*tileSz;

    int x=xl+threadIdx.x/tileSz,
        y=yt+threadIdx.x%tileSz;

    ft cosT, sinT;  // cosine and sine of theta
    ft tileSzSinT, tileSzCosT;

    // for each point (x,y) on the ray is
    // x= t*cosT + (t*cosT+d*sinT)/(t*sinT-d*cosT)*(y-t*sinT);
    // or x = -t*d/(t*sinT-d*cosT) + y*(t*cosT+d*sinT)/(t*sinT-d*cosT);
    ft qe, oc, qa;
    ft bq;
    ft cosB, sinB; //, tanB=t/d;
    ft cosR, sinR;
    ft beamWidth = conf->dSize*conf->effectiveRate;
    ft bw;

    ft dist, weight;
    int temp, imgIdx;
    ft imgt[8];
    ft t, tl, tr, tll, trr;
    int dt, dtl, dtr, dtll, dtrr;
    for(int i=0; i<8; i++) imgt[i]=0;
    __shared__ volatile ft shared[8][ANG_BLK][4*TILE_SZ];
    for(thetaIdx=0; thetaIdx<conf->prjFull;thetaIdx++){

        theta  = thetaIdx *2*PI/conf->prjFull;
        cosT = cos(theta ); sinT = sin(theta );
        tileSzCosT=tileSz*cosT; tileSzSinT=tileSz*sinT;

        // up letf
        qe = (xl-0.5)*sinT - (yt-0.5)*cosT + d;
        oc = (xl-0.5)*cosT + (yt-0.5)*sinT;
        tll = d*oc/qe; trr = tll;

        // up right
        //qe = (xr+0.5)*sinTl - (yt-0.5)*cosTl + d;
        //oc = (xr+0.5)*cosTl + (yt-0.5)*sinTl;
        qe = qe+tileSzSinT; oc = oc+tileSzCosT; t=d*oc/qe;
        tll = min(tll, t); trr=max(trr,t);

        // bottom right
        //qe = (xr+0.5)*sinTl - (yb+0.5)*cosTl + d;
        //oc = (xr+0.5)*cosTl + (yb+0.5)*sinTl;
        qe = qe-tileSzCosT; oc = oc+tileSzSinT; t=d*oc/qe;
        tll = min(tll, t); trr=max(trr,t);

        // bottom left
        //qe = (xl-0.5)*sinTl - (yb+0.5)*cosTl + d;
        //oc = (xl-0.5)*cosTl + (yb+0.5)*sinTl;
        qe = qe-tileSzSinT; oc = oc-tileSzCosT; t=d*oc/qe;
        tll = min(tll, t); trr=max(trr,t);

        // up letf
        qe = (x-0.5)*sinT - (y-0.5)*cosT + d;
        oc = (x-0.5)*cosT + (y-0.5)*sinT;
        tl = d*oc/qe; tr = tl;

        // up right
        //qe = (x+0.5)*sinT - (y-0.5)*cosT + d;
        //oc = (x+0.5)*cosT + (y-0.5)*sinT;
        qe = qe+sinT; oc = oc + cosT; t = d*oc/qe;
        tl = min(tl, t); tr=max(tr,t);

        // bottom right
        //qe = (x+0.5)*sinT - (y+0.5)*cosT + d;
        //oc = (x+0.5)*cosT + (y+0.5)*sinT;
        qe = qe-cosT; oc=oc+sinT; t = d*oc/qe;
        tl = min(tl, t); tr=max(tr,t);

        // bottom left
        //qe = (x-0.5)*sinT - (y+0.5)*cosT + d;
        //oc = (x-0.5)*cosT + (y+0.5)*sinT;
        qe = qe-sinT; oc = oc-cosT; t = d*oc/qe;
        tl = min(tl, t); tr=max(tr,t);

        dtll=max((int)round((tll)/conf->dSize),-(conf->prjWidth-1)/2);
        dtrr=min((int)round((trr)/conf->dSize), (conf->prjWidth-1)/2);

        dtl = max((int)round((tl)/conf->dSize),-(conf->prjWidth-1)/2);
        dtr = min((int)round((tr)/conf->dSize), (conf->prjWidth-1)/2);
        dtl = max(dtl,dtll); dtr=min(dtr,dtrr);

        //qe = d+x*sinT-y*cosT; // positive direction is qo
        qe = qe + sinT/2 +cosT/2;
        //qa = sqrt(2*d*qe-d*d+x*x+y*y);

        __syncthreads();
        for(dt=dtll+threadIdx.x; dt<=dtrr; dt+=blockDim.x){
            temp=thetaIdx;
            shared[0][0][dt-dtll]=temp<conf->np ? sino[temp*conf->prjWidth+dt+pC] : 0;

            temp=(thetaIdx+conf->prjFull/4)%conf->prjFull;
            shared[1][0][dt-dtll]=temp<conf->np ? sino[temp*conf->prjWidth+dt+pC] : 0;

            temp=(thetaIdx+conf->prjFull/2)%conf->prjFull;
            shared[2][0][dt-dtll]=temp<conf->np ? sino[temp*conf->prjWidth+dt+pC] : 0;

            temp=(thetaIdx+3*conf->prjFull/4)%conf->prjFull;
            shared[3][0][dt-dtll]=temp<conf->np ? sino[temp*conf->prjWidth+dt+pC] : 0;

            temp=(3*conf->prjFull/2-thetaIdx)%conf->prjFull;
            shared[4][0][dt-dtll]=temp<conf->np ? sino[temp*conf->prjWidth-dt+pC] : 0;

            temp=(7*conf->prjFull/4-thetaIdx)%conf->prjFull;
            shared[5][0][dt-dtll]=temp<conf->np ? sino[temp*conf->prjWidth-dt+pC] : 0;

            temp=(conf->prjFull-thetaIdx)%conf->prjFull;
            shared[6][0][dt-dtll]=temp<conf->np ? sino[temp*conf->prjWidth-dt+pC] : 0;

            temp=(5*conf->prjFull/4-thetaIdx)%conf->prjFull;
            shared[7][0][dt-dtll]=temp<conf->np ? sino[temp*conf->prjWidth-dt+pC] : 0;
        }
        __syncthreads();

        for(dt=dtl; dt<=dtr; dt++){
            t = dt*conf->dSize;
            bq=sqrt(d*d+t*t);
            cosB=d/bq; sinB=t/bq; //, tanB=t/d;
            cosR=cosB*cosT-sinB*sinT;
            sinR=sinB*cosT+cosB*sinT;
            dist=x*cosR+y*sinR-d*t/bq;
            bw = qe*beamWidth*cosB/d;
            weight=getWeight(dist,bw,cosR,sinR);

            // method provide by the books
            if(FBP) weight = weight*d*d/qe/qe;
            // The one I think should be
            //if(FBP) weight = weight*d/qa;
            //if(FBP) weight = weight*d/qe;

            imgt[0] += weight*shared[0][0][dt-dtll];
            imgt[1] += weight*shared[1][0][dt-dtll];
            imgt[2] += weight*shared[2][0][dt-dtll];
            imgt[3] += weight*shared[3][0][dt-dtll];
            imgt[4] += weight*shared[4][0][dt-dtll];
            imgt[5] += weight*shared[5][0][dt-dtll];
            imgt[6] += weight*shared[6][0][dt-dtll];
            imgt[7] += weight*shared[7][0][dt-dtll];

        }
    }
    if(FBP) for(int i=0; i<8; i++) imgt[i]=imgt[i]*PI/conf->np;
    if(x>(N-1)/2 || y>(N-1)/2) return;
    imgIdx = ( y+N/2)*N+x+N/2; img[imgIdx] = imgt[0]/conf->effectiveRate;
    imgIdx = ( x+N/2)*N-y+N/2; img[imgIdx] = imgt[1]/conf->effectiveRate;
    imgIdx = (-y+N/2)*N-x+N/2; img[imgIdx] = imgt[2]/conf->effectiveRate;
    imgIdx = (-x+N/2)*N+y+N/2; img[imgIdx] = imgt[3]/conf->effectiveRate;
    if(y==0 || y>=x) return;
    imgIdx = (-y+N/2)*N+x+N/2; img[imgIdx] = imgt[4]/conf->effectiveRate;
    imgIdx = ( x+N/2)*N+y+N/2; img[imgIdx] = imgt[5]/conf->effectiveRate;
    imgIdx = ( y+N/2)*N-x+N/2; img[imgIdx] = imgt[6]/conf->effectiveRate;
    imgIdx = (-x+N/2)*N-y+N/2; img[imgIdx] = imgt[7]/conf->effectiveRate;
}

__global__ void rayDrivePar(ft* img, ft* sino){
    //printf("entering rayDrive\n");
    // detector array is of odd size with the center at the middle
    prjConf* conf = &dConf;
    int sinoIdx;
    int thetaIdx = blockIdx.x;
    int tIdx = blockIdx.y*blockDim.x+threadIdx.x;
    int tllIdx = blockIdx.y*blockDim.x;
    int trrIdx = blockIdx.y*blockDim.x+blockDim.x-1;

    if(conf->prjWidth%2==0){
        tIdx++; tllIdx++; trrIdx++;
    }
    //printf("tIdx=%d, thetaIdx=%d\n",tIdx, thetaIdx);

    //if(blockIdx.x==0 && blockIdx.y==0)
    //    printf("gridDim=(%d, %d)\t blockDim=(%d, %d)\n",
    //            gridDim.x, gridDim.y, blockDim.x, blockDim.y);

    ft theta;       // the current angle, range in [0 45]
    ft t;           // position of current detector

    int N =conf->n;// N is of size NxN centering at (N/2,N/2)
    int pC = conf->prjWidth/2;
    ft beamWidth = conf->dSize*conf->effectiveRate;
    ft hbeamW = beamWidth/2;

    __shared__ volatile ft shared[8][2*THRD_SZ];
    // the length should be at least blockDim.x*sqrt(2)*(1+N/2/d)

    theta = thetaIdx*2*PI/conf->prjFull;
    t = (tIdx-pC)*conf->dSize;
    ft tl = t-hbeamW,
       tr = t+hbeamW,
       tll = (tllIdx-pC)*conf->dSize-hbeamW,
       trr = (trrIdx-pC)*conf->dSize+hbeamW;

    ft cosT, sinT;  // cosine and sine of theta
    cosT=cos(theta); sinT=sin(theta);

    // for each point (x,y) on the ray is
    // x= t*cosT + (t*cosT+d*sinT)/(t*sinT-d*cosT)*(y-t*sinT);
    // or x = -t*d/(t*sinT-d*cosT) + y*(t*cosT+d*sinT)/(t*sinT-d*cosT);

    //if(tIdx==pC && thetaIdx==30){
    //    printf("theta=%f,cosT=%f, sinT=%f\n",theta,cosT,sinT);
    //    printf("cosB=%f, sinB=%f\n",cosB,sinB);
    //}

    ft   xl, xll, xr, xrr;
    int dxl,dxll,dxr,dxrr;

    int x,y=(N-1)/2;
    //xc = xc + y*slopeXYc;

    // beamw is based on the position of this pixel
    ft dist, weight;
    int temp;
    ft sinot[8];
    for(int i=0; i<8; i++) sinot[i]=0;
    for(y=(N-1)/2; y>=-(N-1)/2; y--){
        xl = ( tl-sinT *(y+0.5f))/cosT;
        xr = ( tr-sinT *(y-0.5f))/cosT;
        xll= (tll-sinT *(y+0.5f))/cosT;
        xrr= (trr-sinT *(y-0.5f))/cosT;

        dxll=max((int)round(xll),-(N-1)/2);
        dxrr=min((int)round(xrr), (N-1)/2);
        dxl =max((int)round(xl ),-(N-1)/2);
        dxr =min((int)round(xr ), (N-1)/2);
        if(dxl<dxll || dxr>dxrr) printf("rayDrivePar:%d, %d; %d, %d\n",
                dxll,dxl,dxr,dxrr);
        dxl =max(dxl,dxll); dxr=min(dxr,dxrr);

        __syncthreads();
        for(x=dxll+threadIdx.x, temp=threadIdx.x; x<=dxrr;
                x+=blockDim.x, temp+=blockDim.x){
            shared[0][temp] = img[( y+N/2)*N+x+N/2];
            shared[1][temp] = img[( x+N/2)*N-y+N/2];
            shared[2][temp] = img[(-y+N/2)*N-x+N/2];
            shared[3][temp] = img[(-x+N/2)*N+y+N/2];
            shared[4][temp] = img[(-y+N/2)*N+x+N/2];
            shared[5][temp] = img[( x+N/2)*N+y+N/2];
            shared[6][temp] = img[( y+N/2)*N-x+N/2];
            shared[7][temp] = img[(-x+N/2)*N-y+N/2];
        }
        __syncthreads();
        //if(thetaIdx==45 && blockIdx.y==1 && threadIdx.x==0){
        //    printf("\nthetaIdx=%d, t=%f, y=%d, %d->%d\n \t",
        //            thetaIdx,t,y,dxll,dxrr);
        //    for(temp=0; temp<=dxrr-dxll; temp++){
        //        if(shared[1][temp]>0)
        //            printf("%d: %d: %f",temp,temp+dxll,shared[1][temp]);
        //    }
        //}
        for(x=dxl; x<=dxr; x++){
            dist=x*cosT+y*sinT-t;
            weight=getWeight(dist,beamWidth,cosT,sinT);

            sinot[0]+=weight*shared[0][x-dxll];

            //if(thetaIdx==42 && blockIdx.y==4 && threadIdx.x==0){
            //    printf("%d %d %e %e %e\n",y,x,weight, weight*shared[0][x-dxll],sinot[0]);
            //}

            //temp=thetaIdx+conf->prjFull/4;
            sinot[1]+=weight*shared[1][x-dxll]; //img[imgIdx];
            //temp+=conf->prjFull/4;
            sinot[2]+=weight*shared[2][x-dxll]; //img[imgIdx];
            //temp+=conf->prjFull/4;
            sinot[3]+=weight*shared[3][x-dxll]; //img[imgIdx];
            //temp=conf->prjFull/2-thetaIdx;
            sinot[4]+=weight*shared[4][x-dxll]; //img[imgIdx];
            //temp=3*conf->prjFull/4-thetaIdx;
            sinot[5]+=weight*shared[5][x-dxll]; //img[imgIdx];
            //temp=conf->prjFull-thetaIdx;
            sinot[6]+=weight*shared[6][x-dxll]; //img[imgIdx];
            //temp=conf->prjFull/4-thetaIdx;
            sinot[7]+=weight*shared[7][x-dxll]; //img[imgIdx];
        }
    }

    if(tIdx>pC) return;

    if(thetaIdx<conf->np){
        sinoIdx=thetaIdx*conf->prjWidth;
        sino[sinoIdx+tIdx]=sinot[0]/conf->effectiveRate;
        sino[sinoIdx+2*pC-tIdx]=sinot[2]/conf->effectiveRate;
    }

    temp = thetaIdx+conf->prjFull/4;
    if(temp<conf->np){
        sinoIdx = temp*conf->prjWidth;
        sino[sinoIdx+tIdx]=sinot[1]/conf->effectiveRate;
        sino[sinoIdx+2*pC-tIdx]=sinot[3]/conf->effectiveRate;
    }

    temp = thetaIdx+conf->prjFull/2;
    if(temp<conf->np){
        sinoIdx = temp*conf->prjWidth;
        sino[sinoIdx+tIdx]=sinot[2]/conf->effectiveRate;
        sino[sinoIdx+2*pC-tIdx]=sinot[0]/conf->effectiveRate;
    }

    temp = thetaIdx+3*conf->prjFull/4;
    if(temp<conf->np){
        sinoIdx = temp*conf->prjWidth;
        sino[sinoIdx+tIdx]=sinot[3]/conf->effectiveRate;
        sino[sinoIdx+2*pC-tIdx]=sinot[1]/conf->effectiveRate;
    }

    if(thetaIdx>0 && thetaIdx<conf->prjFull*0.125f){
        tIdx = 2*pC-tIdx;

        temp = conf->prjFull/2-thetaIdx;
        if(temp<conf->np){
            sinoIdx = temp*conf->prjWidth;
            sino[sinoIdx+tIdx]=sinot[4]/conf->effectiveRate;
            sino[sinoIdx+2*pC-tIdx]=sinot[6]/conf->effectiveRate;
        }

        temp = 3*conf->prjFull/4-thetaIdx;
        if(temp<conf->np){
            sinoIdx = temp*conf->prjWidth;
            sino[sinoIdx+tIdx]=sinot[5]/conf->effectiveRate;
            sino[sinoIdx+2*pC-tIdx]=sinot[7]/conf->effectiveRate;
        }

        temp = conf->prjFull-thetaIdx;
        if(temp<conf->np){
            sinoIdx = temp*conf->prjWidth;
            sino[sinoIdx+tIdx]=sinot[6]/conf->effectiveRate;
            sino[sinoIdx+2*pC-tIdx]=sinot[4]/conf->effectiveRate;
        }

        temp = conf->prjFull/4-thetaIdx;
        if(temp<conf->np){
            sinoIdx = temp*conf->prjWidth;
            sino[sinoIdx+tIdx]=sinot[7]/conf->effectiveRate;
            sino[sinoIdx+2*pC-tIdx]=sinot[5]/conf->effectiveRate;
        }
    }
}

__global__ void rayDriveFan(ft* img, ft* sino){
    // printf("entering rayDrive\n");
    // detector array is of odd size with the center at the middle
    prjConf* conf = &dConf;
    int sinoIdx;
    int thetaIdx = blockIdx.x;
    int tIdx   = blockIdx.y*blockDim.x+threadIdx.x;
    int tllIdx = blockIdx.y*blockDim.x;
    int trrIdx = blockIdx.y*blockDim.x+blockDim.x-1;

    if(conf->prjWidth%2==0){
        tIdx++; trrIdx++; tllIdx++;
    }
    //printf("tIdx=%d, thetaIdx=%d\n",tIdx, thetaIdx);

#if DEBUG
    if(blockIdx.x==0 && blockIdx.y==0 && threadIdx.x==0)
        printf("gridDim=(%d, %d)\t blockDim=(%d, %d)\n",
                gridDim.x, gridDim.y, blockDim.x, blockDim.y);
#endif

    ft theta;       // the current angle, range in [0 45]
    ft t;           // position of current detector
    ft d;           // the distance from rotating center to X-ray source in pixel

    int N =conf->n;// N is of size NxN centering at (N/2,N/2)
    int pC = conf->prjWidth/2;
    ft beamWidth = conf->dSize*conf->effectiveRate;
    ft hbeamW = beamWidth/2;

    // adding shared memory improves performance from 2.1s to 1.3s in 
    // Quadro 600
    // improves from 0.2s to 0.08s in Tesla K20
    // improves from 0.2s to 0.18s in Tesla K10

    __shared__ volatile ft shared[8][2*THRD_SZ];
    // the length should be at least blockDim.x*sqrt(2)*(1+N/2/d)

    theta = thetaIdx*2*PI/conf->prjFull;
    t = (tIdx-pC)*conf->dSize;
    ft tl = t-hbeamW,
       tr = t+hbeamW,
       tll = (tllIdx-pC)*conf->dSize-hbeamW,
       trr = (trrIdx-pC)*conf->dSize+hbeamW;
    d = conf->d;

    ft cosT, sinT;  // cosine and sine of theta
    cosT=cos(theta); sinT=sin(theta);

    // for each point (x,y) on the ray is
    // x= t*cosT + (t*cosT+d*sinT)/(t*sinT-d*cosT)*(y-t*sinT);
    // or x = -t*d/(t*sinT-d*cosT) + y*(t*cosT+d*sinT)/(t*sinT-d*cosT);
    ft bq=sqrt(d*d+t*t);
    ft cosB=d/bq, sinB=t/bq; //, tanB=t/d;
    ft cosR=cosB*cosT-sinB*sinT; // cosR and sinR are based on t
    ft sinR=sinB*cosT+cosB*sinT;
    ft beamwidthCosB=beamWidth*cosB;

    //if(tIdx==pC && thetaIdx==30){
    //    printf("theta=%f,cosT=%f, sinT=%f\n",theta,cosT,sinT);
    //    printf("cosB=%f, sinB=%f\n",cosB,sinB);
    //}

    ft dtl=d*tl, dtll=d*tll, dtr=d*tr, dtrr=d*trr;
    ft QxBxl = tl *cosT+d*sinT, QxBxr = tr *cosT+d*sinT,
       QxBxll= tll*cosT+d*sinT, QxBxrr= trr*cosT+d*sinT;

    ft QyByl =-tl *sinT+d*cosT, QyByr =-tr *sinT+d*cosT,
       QyByll=-tll*sinT+d*cosT, QyByrr=-trr*sinT+d*cosT;

    ft   xl, xll, xr, xrr;
    int dxl,dxll,dxr,dxrr;

    int x,y=(N-1)/2;
    //xc = xc + y*slopeXYc;

    // beamw is based on the position of this pixel
    ft bw, dist, weight;
    int temp;
    ft sinot[8];
    for(int i=0; i<8; i++) sinot[i]=0;
    for(y=(N-1)/2; y>=-(N-1)/2; y--){
        if(QxBxl>0) xl = (dtl - QxBxl *(y+0.5f))/QyByl;
        else xl = (dtl -QxBxl *(y-0.5f))/QyByl;
        if(QxBxr>0) xr = (dtr - QxBxr *(y-0.5f))/QyByr;
        else xr = (dtr - QxBxr *(y+0.5f))/QyByr;

        if(QxBxll>0) xll= (dtll- QxBxll*(y+0.5f))/QyByll;
        else xll= (dtll-QxBxll*(y-0.5f))/QyByll;
        if(QxBxrr>0) xrr= (dtrr- QxBxrr*(y-0.5f))/QyByrr;
        else xrr= (dtrr- QxBxrr*(y+0.5f))/QyByrr;

        dxll=max((int)round(xll),-(N-1)/2);
        dxrr=min((int)round(xrr), (N-1)/2);
        dxl =max((int)round(xl ),-(N-1)/2);
        dxr =min((int)round(xr ), (N-1)/2);
        dxl =max(dxl,dxll); dxr=min(dxr,dxrr);

        __syncthreads();
        for(x=dxll+threadIdx.x, temp=threadIdx.x; x<=dxrr;
                x+=blockDim.x, temp+=blockDim.x){
            shared[0][temp] = img[( y+N/2)*N+x+N/2];
            shared[1][temp] = img[( x+N/2)*N-y+N/2];
            shared[2][temp] = img[(-y+N/2)*N-x+N/2];
            shared[3][temp] = img[(-x+N/2)*N+y+N/2];
            shared[4][temp] = img[(-y+N/2)*N+x+N/2];
            shared[5][temp] = img[( x+N/2)*N+y+N/2];
            shared[6][temp] = img[( y+N/2)*N-x+N/2];
            shared[7][temp] = img[(-x+N/2)*N-y+N/2];
        }
        __syncthreads();
        //if(thetaIdx==45 && blockIdx.y==1 && threadIdx.x==0){
        //    printf("\nthetaIdx=%d, t=%f, y=%d, %d->%d\n \t",
        //            thetaIdx,t,y,dxll,dxrr);
        //    for(temp=0; temp<=dxrr-dxll; temp++){
        //        if(shared[1][temp]>0)
        //            printf("%d: %d: %f",temp,temp+dxll,shared[1][temp]);
        //    }
        //}
        for(x=dxl; x<=dxr; x++){

            dist=x*cosR+y*sinR-d*t/bq;
            bw=beamwidthCosB + (x*sinT-y*cosT)*beamwidthCosB/d;
            weight=getWeight(dist,bw,cosR,sinR);

            sinot[0]+=weight*shared[0][x-dxll];

            //if(thetaIdx==42 && blockIdx.y==4 && threadIdx.x==0){
            //    printf("%d %d %e %e %e\n",y,x,weight, weight*shared[0][x-dxll],sinot[0]);
            //}

            //temp=thetaIdx+conf->prjFull/4;
            sinot[1]+=weight*shared[1][x-dxll]; //img[imgIdx];
            //temp+=conf->prjFull/4;
            sinot[2]+=weight*shared[2][x-dxll]; //img[imgIdx];
            //temp+=conf->prjFull/4;
            sinot[3]+=weight*shared[3][x-dxll]; //img[imgIdx];
            //temp=conf->prjFull/2-thetaIdx;
            sinot[4]+=weight*shared[4][x-dxll]; //img[imgIdx];
            //temp=3*conf->prjFull/4-thetaIdx;
            sinot[5]+=weight*shared[5][x-dxll]; //img[imgIdx];
            //temp=conf->prjFull-thetaIdx;
            sinot[6]+=weight*shared[6][x-dxll]; //img[imgIdx];
            //temp=conf->prjFull/4-thetaIdx;
            sinot[7]+=weight*shared[7][x-dxll]; //img[imgIdx];
        }
    }

    if(tIdx>=conf->prjWidth) return;

    if(thetaIdx<conf->np){
        sinoIdx=thetaIdx*conf->prjWidth+tIdx;
        sino[sinoIdx]=sinot[0]/conf->effectiveRate;
    }

    temp = thetaIdx+conf->prjFull/4;
    if(temp<conf->np){
        sino[temp*conf->prjWidth+tIdx]=sinot[1]/conf->effectiveRate;
    }

    temp = thetaIdx+conf->prjFull/2;
    if(temp<conf->np){
        sino[temp*conf->prjWidth+tIdx]=sinot[2]/conf->effectiveRate;
    }

    temp = thetaIdx+3*conf->prjFull/4;
    if(temp<conf->np){
        sino[temp*conf->prjWidth+tIdx]=sinot[3]/conf->effectiveRate;
    }

    if(thetaIdx>0 && thetaIdx<conf->prjFull*0.125f){
        tIdx = 2*pC-tIdx;

        temp = conf->prjFull/2-thetaIdx;
        if(temp<conf->np)
            sino[temp*conf->prjWidth+tIdx]=sinot[4]/conf->effectiveRate;

        temp = 3*conf->prjFull/4-thetaIdx;
        if(temp<conf->np)
            sino[temp*conf->prjWidth+tIdx]=sinot[5]/conf->effectiveRate;

        temp = conf->prjFull-thetaIdx;
        if(temp<conf->np)
            sino[temp*conf->prjWidth+tIdx]=sinot[6]/conf->effectiveRate;

        temp = conf->prjFull/4-thetaIdx;
        if(temp<conf->np)
            sino[temp*conf->prjWidth+tIdx]=sinot[7]/conf->effectiveRate;
    }
}

#ifdef __cplusplus
extern "C"
#endif
void setup(int n, int prjWidth, int np, int prjFull, ft dSize, ft 
        effectiveRate, ft d){
    config.n=n; config.prjWidth=prjWidth;
    config.np=np; config.prjFull=prjFull;
    config.dSize=dSize; config.effectiveRate=effectiveRate;
    config.d=d;

    config.imgSize=config.n*config.n;
    config.sinoSize=config.prjWidth*config.np;

    if(config.d>0){
        if(pConf->prjWidth%2==0)
            fGrid = dim3(
                min(pConf->np, pConf->prjFull/8+1),
                (pConf->prjWidth-1+THRD_SZ-1)/THRD_SZ
                );
        else
            fGrid = dim3(
                min(pConf->np, pConf->prjFull/8+1),
                (pConf->prjWidth+THRD_SZ-1)/THRD_SZ
                );
        fThread = dim3(THRD_SZ,LYR_BLK);

        // use the last block to make frame zero.
        int temp = ((pConf->n+1)/2+TILE_SZ-1)/TILE_SZ;
        bGrid = dim3((1+temp)*temp/2);
        bThread = dim3(TILE_SZ*TILE_SZ, ANG_BLK);
    }else{
        fGrid = dim3(
                min(pConf->np, pConf->prjFull/8+1),
                ((pConf->prjWidth+1)/2+THRD_SZ-1)/THRD_SZ
                );
        fThread = dim3(THRD_SZ,LYR_BLK);

        int temp = ((pConf->n+1)/2+TILE_SZ-1)/TILE_SZ;
        bGrid = dim3((1+temp)*temp/2);
        bThread = dim3(TILE_SZ*TILE_SZ, ANG_BLK);
    }

    hipDeviceReset();
    HANDLE_ERROR(hipMalloc((void**)&dev_img,pConf->imgSize*sizeof(ft)));
    HANDLE_ERROR(hipMalloc((void**)&dev_sino,pConf->prjFull*pConf->prjWidth*sizeof(ft)));
    HANDLE_ERROR(hipMemcpyToSymbol(HIP_SYMBOL( dConf), pConf, sizeof(prjConf)) );

#if EXE_TIME
    // start the timers
    HANDLE_ERROR( hipEventCreate( &start ) );
    HANDLE_ERROR( hipEventCreate( &stop ) );
#endif
#if DEBUG
    printf("fGrid=(%d,%d), fThread=(%d,%d), bGrid=(%d,%d), bThread=(%d,%d)\n",
            fGrid.x,fGrid.y,fThread.x,fThread.y,
            bGrid.x,bGrid.y,bThread.x,bThread.y);
    printf("setup done\n");
#endif
}

void showSetup(){
    printf("config.n=%d\n",config.n);
    printf("config.prjWidth=%d\n",config.prjWidth);
    printf("config.np=%d\n",config.np);
    printf("config.prjFull=%d\n",config.prjFull);
    printf("config.dSize=%g\n",config.dSize);
    printf("config.effectiveRate=%g\n",config.effectiveRate);
    printf("config.d=%g\n",config.d);
}

#ifdef __cplusplus
extern "C"
#endif
void cleanUp(){
#if EXE_TIME
    HANDLE_ERROR( hipEventDestroy( start ) );
    HANDLE_ERROR( hipEventDestroy( stop ) );
#endif
    HANDLE_ERROR( hipFree( dev_img ) );
    HANDLE_ERROR( hipFree( dev_sino ) );
    hipDeviceReset();
}

#ifdef __cplusplus
extern "C"
#endif
int gpuPrj(ft* img, ft* sino, char cmd){
#if EXE_PROF
    hipProfilerStart();
#endif

#if EXE_TIME
    // start the timers
    HANDLE_ERROR( hipEventRecord( start, 0 ) );
    HANDLE_ERROR( hipEventSynchronize( start ) );
#endif

    if(cmd & FWD_BIT){
#if DEBUG
        printf("Forward projecting ...\n");
#endif
        HANDLE_ERROR(hipMemcpy(dev_img, img, pConf->imgSize*sizeof(ft), 
                    hipMemcpyHostToDevice ) );
        if(pConf->d>0){
#if DEBUG
        printf("Image copied to device ...\n");
        printf("calling rayDriveFan ...\n");
#endif
            rayDriveFan<<<fGrid,fThread>>>(dev_img, dev_sino);
        }else
            rayDrivePar<<<fGrid,fThread>>>(dev_img, dev_sino);
        HANDLE_ERROR( hipMemcpy( sino, dev_sino, pConf->sinoSize*sizeof(ft),
                    hipMemcpyDeviceToHost ) );

        if(pConf->prjWidth%2==0)
            for(int i=0,idx=0; i<pConf->np; i++,idx+=pConf->prjWidth)
                sino[idx]=0;
        
    }else if(cmd & BWD_BIT){
#if DEBUG
        printf("Backward projecting ...\n");
#endif
#if DEBUG
        {
            FILE* f = fopen("sinogram_0.data","wb");
            fwrite(sino, sizeof(ft), config.sinoSize, f);
            fclose(f);
        }
#endif
        HANDLE_ERROR(hipMemcpy(dev_sino,sino,pConf->sinoSize*sizeof(ft),
                    hipMemcpyHostToDevice ) );
        if(pConf->d>0){
            pixelDriveFan<<<bGrid,bThread>>>(dev_img, dev_sino,0);
        }else
            pixelDrivePar<<<bGrid,bThread>>>(dev_img, dev_sino,0);

        HANDLE_ERROR( hipMemcpy( img, dev_img, pConf->imgSize*sizeof(ft),
                    hipMemcpyDeviceToHost ) );
        if(pConf->n%2==0){
            for(int i=0,idx=0; i<pConf->n; i++,idx+=pConf->n){
                img[i]=0; img[idx]=0;
            }
        }
    }else if(cmd & FBP_BIT){
#if DEBUG
        printf("Filtered Backprojecting ...\n");
#endif
        ft* pSino = (ft*) calloc(pConf->sinoSize,sizeof(ft));
        ft bq;
        int pC = pConf->prjWidth/2;

#if DEBUG
        FILE* f = fopen("sinogram_0.data","wb");
        fwrite(sino, sizeof(ft), config.sinoSize, f);
        fclose(f);
#endif

        if(pConf->d>0){
#if DEBUG
            printf("reconstructing by FBP (fan beam) ... \n");
#endif
            for(int j=0; j<(pConf->prjWidth+1)/2; j++){
                bq = sqrt(pConf->d*pConf->d + j*j*pConf->dSize*pConf->dSize);
                for(int i=0, idx1=pC-j, idx2=pC+j; i<pConf->np;
                        i++, idx1+=pConf->prjWidth, idx2+=pConf->prjWidth){
                    pSino[idx1]=sino[idx1]*pConf->d / bq;
                    pSino[idx2]=sino[idx2]*pConf->d / bq;
                }
            }
            if(pConf->prjWidth%2==0){
                bq = sqrt(pConf->d*pConf->d + pC*pC*pConf->dSize*pConf->dSize);
                for(int i=0, idx1=0; i<pConf->np;
                        i++, idx1+=pConf->prjWidth){
                    pSino[idx1]=sino[idx1]*pConf->d / bq;
                }
            }
        }else{
#if DEBUG
            printf("reconstructing by FBP (parallel beam) ... \n");
#endif
            memcpy(pSino,sino,pConf->sinoSize*sizeof(ft));
        }

#if DEBUG
        f = fopen("sinogram_1.data","wb");
        fwrite(pSino, sizeof(ft), config.sinoSize, f);
        fclose(f);
#endif

        for(int i=0; i<pConf->np; i++){
            rampFilter(pSino+i*pConf->prjWidth, pConf->prjWidth, pConf->dSize);
        }

#if DEBUG
        f = fopen("sinogram_2.data","wb");
        fwrite(pSino, sizeof(ft), config.sinoSize, f);
        fclose(f);
#endif
        HANDLE_ERROR(hipMemcpy(dev_sino,pSino,pConf->sinoSize*sizeof(ft),
                    hipMemcpyHostToDevice ) );
        if(pConf->d>0)
            pixelDriveFan<<<bGrid,bThread>>>(dev_img, dev_sino,1);
        else
            pixelDrivePar<<<bGrid,bThread>>>(dev_img, dev_sino,1);
        HANDLE_ERROR( hipMemcpy( img, dev_img, pConf->imgSize*sizeof(ft),
                    hipMemcpyDeviceToHost ) );

        if(pConf->n%2==0){
            for(int i=0,idx=0; i<pConf->n; i++,idx+=pConf->n){
                img[i]=0; img[idx]=0;
            }
        }
        free(pSino);
    }


#if EXE_PROF
    hipProfilerStop();
#endif

#if EXE_TIME
    float elapsedTime;
    HANDLE_ERROR( hipEventRecord( stop, 0 ) );
    HANDLE_ERROR( hipEventSynchronize( stop ) );
    HANDLE_ERROR( hipEventElapsedTime( &elapsedTime,
                start, stop ) );
    printf( "Time taken:  %3.1f ms\n", elapsedTime );
#endif
    //FILE* f = fopen("sinogram.data","wb");
    //fwrite(sino, sizeof(ft), pConf->sinoSize, f);
    //fclose(f);
    return 0;
}

int forwardTest( void ) {
    ft* img = (ft*) malloc(config.imgSize*sizeof(ft));
    ft *sino = (ft *) malloc(config.sinoSize*sizeof(ft));
    int offset;
    int YC = config.n/2, XC = config.n/2;
    for(int i=0; i < config.n; i++){
        for(int j=0; j < config.n; j++){
            offset = i*config.n+j;
            if(((i-YC-0.02*config.n)*(i-YC-0.02*config.n)+(j-XC-0)*(j-XC-0)<=(0.32*config.n)*(0.32*config.n))
                    && ((i-YC-0.12*config.n)*(i-YC-0.12*config.n)+
                        (j-XC-0.12*config.n)*(j-XC-0.12*config.n)>=(0.088*config.n)*(0.088*config.n))
              ){
                img[offset]=1;
            }else
                img[offset]=0;
            //            if(i<5 && j < 5) img[i][j]=1;
        }
    }
#if DEBUG
    FILE* f = fopen("img.data","w");
    fwrite(img,sizeof(ft), pConf->imgSize,f);
    fclose(f);
#endif
#if SHOWIMG
    show_img(img,config.n,config.n);
#endif
    gpuPrj(img, sino, RENEW_MEM | FWD_BIT);
#if DEBUG
    f = fopen("sinogram.data","wb");
    fwrite(sino, sizeof(ft), config.sinoSize, f);
    fclose(f);
#endif
#if SHOWIMG
    show_img(sino,config.prjWidth,config.np);
#endif
    free(img); free(sino);
    return 0;
}

int backwardTest( void ) {
    ft* img = (ft*) malloc(config.imgSize*sizeof(ft));
    ft *sino = (ft *) malloc(config.sinoSize*sizeof(ft));

#if DEBUG
    FILE* f;
    f = fopen("sinogram_0.data","rb");
    if(f==NULL || !fread(sino,sizeof(ft),config.sinoSize,f)){
        perror("cannot read from sinogram.data\n");
        exit(0);
    }
    fclose(f);
#endif

    ft tempI;
    tempI=rand()%config.np;
    tempI=0;
    for(int i=0; i < config.np; i++){
        for(int j=0; j < config.prjWidth; j++){
            tempI=sino[i*config.prjWidth+j]>tempI? sino[i*config.prjWidth+j] : tempI;
        }
    }
    for(int i=0; i < config.np; i++){
        for(int j=0; j < config.prjWidth; j++){
#if SHOWIMG
            int offset = i*config.prjWidth+j;
            if(i==tempI*0 && abs(j-config.prjWidth/2)<=150){
                if(offset%15<6) sino[offset]=1;
                else sino[offset]=0;
            }else
                sino[offset]=0;
            tempI=1;
#endif
        }
    }
#if SHOWIMG
    show_img(sino,config.np,config.prjWidth);
#endif

    gpuPrj(img, sino, BWD_BIT);

#if DEBUG
    f = fopen("reImg.data","wb");
    fwrite(img,sizeof(ft),config.imgSize,f);
    fclose(f);
#endif
#if SHOWIMG
    show_img(img,config.n,config.n);
#endif
    free(sino); free(img);
    return 0;
}

void FBP(char* filename, int idx){
    FILE* f = fopen(filename,"r");
    int len = 100,N,np;
    float widthDetector,angle,maxVal,dist;
    ft *sino,*img;
    char str[len];
    if(f!=NULL){
        fseek(f,-400,SEEK_END);
        while(!strstr(fgets(str,len,f),"FAN_BEAM_PARAMETERS"));
        fscanf(f,"%f",&dist); printf("dist=%f\n",dist);
        rewind(f);
        fgets(str,len,f); printf("%s",str);
        fscanf(f,"%d",&N); printf("N=%d\n",N);
        fscanf(f,"%d",&np); printf("np=%d\n",np);
        fscanf(f,"%f",&widthDetector); fscanf(f,"%f",&widthDetector);
        printf("widthDetector=%f\n",widthDetector);
        fscanf(f,"%f",&angle); fscanf(f,"%f",&angle);
        printf("angle=%f\n",angle);
        fscanf(f,"%f",&maxVal); fscanf(f,"%f",&maxVal);
        printf("maxVal=%f\n",maxVal);

        int imgSize = (1 << ((int)floor(log2(N*1.0))));
        int prjFull = (int)(np/angle*360);

        ft distOffset = 0;
        int offset = 11;
        int center = N/2+offset;
        int half = MIN(center,N-center),l;
        
        sino = (ft *) malloc(np*N*sizeof(ft));
        img = (ft*) malloc(imgSize*imgSize*sizeof(ft));
        setup(imgSize,2*half,np,prjFull,((double)imgSize)/N,1,
                (dist+distOffset)*imgSize/widthDetector);
        showSetup();

        ft temp;
        long pos = ftell(f);

        //for(offset=10; offset<15; offset++)
        for(int k=0; k<10; k++){
            if(k==0){
                fseek(f,pos,SEEK_SET);
                printf("offset = %d\n",offset);
            }
            center = N/2+offset;
            half = MIN(center,N-center);
            for(int i=0; i < np; i++)
                for(int j=0; j < N; j++){
                    fscanf(f,"%f",&temp);
                    if(temp<=0){
                        sprintf(str,"FBP: get strange measurement %f\n",temp);
                        perror(str);
                    }

                    l = j-(center-half);
                    if(l>=0 && l<config.prjWidth){
                        if(temp==0)
                            sino[i*config.prjWidth+l]=sino[i*config.prjWidth+l-1];
                        else
                            sino[i*config.prjWidth+l] = log10(maxVal/temp);
                    }
                    //sino[i*config.prjWidth+l]=temp;
                }
            if(idx==-1 || k==idx){
#if SHOWIMG
                //show_img(sino,config.prjWidth,config.np);
#endif
                for(distOffset=-400; distOffset<=200; distOffset+=20){
                    printf("FBP: %f\n", distOffset);
                    setup(imgSize,2*half,np,prjFull,((double)imgSize)/N,1,
                            (dist+distOffset)*imgSize/widthDetector);
                    showSetup();

                    gpuPrj(img, sino, FBP_BIT);
#if SHOWIMG
                    show_img(img,config.n,config.n,0);
#endif
                }
#if DEBUG
                //FILE* ff = fopen("img.data","w");
                //fwrite(img,sizeof(ft), config.imgSize,ff);
                //fclose(ff);
                //ff = fopen("sino.data","w");
                //fwrite(sino,sizeof(ft), config.sinoSize,ff);
                //fclose(ff);
#endif
            }
        }

        free(sino); free(img);
    }
    fclose(f);
}

int main(int argc, char *argv[]){
#if SHOWIMG
    CPUBitmap temp(1,1);
    temp.init();
#endif

    if(argc==1){
        int N=512;
        setup(N,N,360,360,1,1,0*N);
        forwardTest();
        //backwardTest();
        cleanUp();
    }else if(argc>=2){
        char* filename = argv[1];
        if(argc==2) FBP(filename,-1);
        else FBP(filename,atoi(argv[2]));
    }
}

